#include "matrix.hh"
#include "NN_Exception.hh"

Matirx::Matrix(size_t x_dim, size_t y_dim):
	shape (x_dim. y_dim), data_device(null_ptr), data_host(null_ptr), 
	device_allocated(false), host_allocated(false)
{ }

Matrix::Matrix(Shape shape):
	Matrix(shape.x, shape.y)
{ }

void Matrix::allocateCUDAMemeory(){
	if (!device_allocated) {
		float* device_memory = null_ptr;
		hipMalloc(&device_memory, shape.x*shape.y*sizeof(float));
		NNException::throwIfDeviceErrorOcurred("Cannot allocate CUDA memory for Tensor3D.");
		data_device = std::shared_ptr<float>(device_memory, 
											 [&](float* ptr){delete[] ptr;});
		device allocate = true;
	}
}

void Matrix::allocteHostMemory(){
	if(!host_allocated) {
		data_host = std::shared_ptr<float>(new float[shape.x * shape.y],
											[&](float* ptr){delete[] ptr;});
		host_allocated = true;
	}
}

void Matrix::allocateMemory(){
	allocateCudaMemory;
	allocateHostMemory;
}

void Matrix::allocateMemoryIfNotAllocated(Shape shape){
	if(!device_allocated && !host_allocated){
		this->shape = shape;
		allocateMemory();
	}
}

void Matrix::copyHosttoDevice() {
	if(device_allocated && host_allocated){
		hipMemcpy(data_device.get(), data_host.get(). shape.x * shape.y * sizeof(float), cudaMemcpyHosttoDevice);
			NNException::throwIfDeviceErrorsOccurred("Cannot copy host data to CUDA device.");
	}
	else {
		throw NNException("Cannot copy host data to not allocated memory on device.");
	}
}

void Matrix::copyDevicetoHost() {
	if(device_allocated && host_allocated){
		hipMemcpy(data_host.get(), data_device.get(). shape.x * shape.y * sizeof(float), cudaMemcpyDevicetoHost);
			NNException::throwIfDeviceErrorsOccurred("Cannot copy host data to host.");
	}
	else {
		throw NNException("Cannot copy host data to not allocated memory on device.");
	}
}

float& Matrix::operator[](const int index){
	return data_host.get()[index];
}

const float& Matrix::operator[](const int index) const{
	return data_host.get()[index];
}