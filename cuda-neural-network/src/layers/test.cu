#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <stdlib.h>
#include "softmax_activation.hh"

__global__ void softmax(float* input, float* buffer, float* max_num, float* row_sum, int Z_x_dim, int Z_y_dim) {
  	assert(input);

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d \n", idx);
	//int col = blockIdx.x * blockDim.x + threadIdx.x;
  	// assert(input_len >= 0);  Not needed
	for (int i = 0; i < Z_y_dim; i++) {
		max_num[i] = -INFINITY;
	}

	if (idx < Z_y_dim ) {
		for (int i = 0; i < Z_x_dim; i++){ 
			if(max_num[idx] < input[idx * Z_x_dim + i] ){
				max_num[idx] = input[idx * Z_x_dim + i];
			}
			
		}
	}
	__syncthreads();

	if(idx < Z_x_dim){
		for(size_t i = 0; i < Z_x_dim; i++)
			buffer[idx * Z_y_dim + i] = expf(input[idx * Z_y_dim + i] - max_num[idx]);
	}

	if(idx < Z_x_dim){
		for(size_t i = 0; i < Z_x_dim; i++)
			row_sum[idx] += buffer[idx * Z_y_dim + i];
		
	}
	if(idx < Z_x_dim){
		for(size_t i = 0; i < Z_x_dim; i++)
			buffer[idx * Z_y_dim + i] = buffer[idx * Z_y_dim + i] / row_sum[idx];
	}


}

int main()
{
	
	float Z[25*25];
	int i ;
 	for( i = 0; i < 25*25; ++i){
		Z[i] = i+1;
    }

  	float buffer[25*25];

 	float *max_num = new float[25];
 	float *row_sum = new float[25];

 	float *Z_d, *buffer_d, *max_num_d, *row_sum_d;
 	hipMalloc((void **)&Z_d, 25*25*sizeof(float));
 	hipMalloc((void **)&buffer_d, 25*25*sizeof(float));
 	hipMalloc((void **)&max_num_d, 25*sizeof(float));
 	hipMalloc((void **)&row_sum_d, 25*sizeof(float));

 	hipMemcpy(Z_d, Z, 25*25*sizeof(float), hipMemcpyHostToDevice);
 	hipMemcpy(buffer_d, buffer, 25*25*sizeof(float), hipMemcpyHostToDevice);
 	hipMemcpy(max_num_d, max_num, 25*sizeof(float), hipMemcpyHostToDevice);
 	hipMemcpy(row_sum_d, row_sum, 25*sizeof(float), hipMemcpyHostToDevice);

 	dim3 block_size(64);
	dim3 num_of_blocks((25 * 25 + block_size.x - 1) / block_size.x);

 	softmax<<<num_of_blocks, block_size>>>(Z_d, buffer_d, max_num_d, row_sum_d, 25, 25);
	
	hipMemcpy(buffer, buffer_d, 25*25*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(max_num, max_num_d, 25*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(row_sum, row_sum_d, 25*sizeof(float), hipMemcpyDeviceToHost);
 	std::cout<<"buffer"<<std::endl;
 	
	for(i = 0; i < 25*25; i++){
    	std::cout<<buffer[i]<<std::endl;
	}
	std::cout<<"max num"<<std::endl;
 	
	for(i = 0; i < 25; i++){
    	std::cout<<max_num[i]<<std::endl;
	}
	std::cout<<"row sum"<<std::endl;
 	
	for(i = 0; i < 25; i++){
    	std::cout<<row_sum[i]<<std::endl;
	}

 	return 0;

}