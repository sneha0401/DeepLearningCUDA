#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <stdlib.h>
#include "softmax_activation.hh"

__global__ void softmax(float* input, float* buffer, float* max_num, int Z_x_dim, int Z_y_dim) {
  	assert(input);

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	//int col = blockIdx.x * blockDim.x + threadIdx.x;
  	// assert(input_len >= 0);  Not needed
	for (int i = 0; i < Z_y_dim; i++) {
		max_num[i] = -INFINITY;
	}

	if (row < Z_y_dim ) {
		for (int i = 0; i < Z_x_dim; i++) {
			if(max_num[i] < input[row * Z_x_dim + i] ){
				max_num[row] = input[row * Z_x_dim + i];
			}
		}
	}
}
int main()
{
	
	float Z[25][25], i ,j;
 	for( i = 0; i < 25; ++i){
  		for( j = 0;  j < 25; ++j){
     		Z[i][j] = rand();
     	}
  	}

  	float buffer[25][25];
 	
 	for( i = 0; i < 25; ++i){
 		for( j = 0; j < 25; ++j)
    		std::cout<<Z[i][j]<<'\t';
   		std::cout<<'\n';
 	}
 	float *max_num = new float[25];

 	softmax<<<2, 20>>>(Z, buffer, max_num, 25, 25);
 	
 	for(i = 0; i < 10; i++){
 		for( j = 0; j < 3; ++j)
    		std::cout<<buffer[i][j]<<'\t';
	}
 	return 0;

}