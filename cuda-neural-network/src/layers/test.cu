#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <stdlib.h>
#include "softmax_activation.hh"

__global__ void softmax(float* input, float* buffer, float* max_num, int Z_x_dim, int Z_y_dim) {
  	assert(input);

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	//int col = blockIdx.x * blockDim.x + threadIdx.x;
  	// assert(input_len >= 0);  Not needed
	for (int i = 0; i < Z_y_dim; i++) {
		max_num[i] = -INFINITY;
	}

	if (row < Z_y_dim ) {
		for (int i = 0; i < Z_x_dim; i++) {
			if(max_num[i] < input[row * Z_x_dim + i] ){
				max_num[row] = input[row * Z_x_dim + i];
			}
		}
	}
}
int main()
{
	
	float Z[25][25];
	int i ,j;
 	for( i = 0; i < 25*25; ++i){
 		for( j =0; j < 25; ++j){
  			Z[i][j] = rand();
    	}
    }

  	float buffer[25][25];

 	for( i = 0; i < 25*25; ++i){
 		for( j =0; j < 25; ++j){
  			buffer[i][j] = rand();
    	}
    }

 	float *max_num = new float[25];

 	float *Z_d, *buffer_d, *max_num_d;
 	hipMalloc((void **)&Z_d, 25*25*sizeof(float));
 	hipMalloc((void **)&buffer_d, 25*25*sizeof(float));
 	hipMalloc((void **)&max_num_d, 25*sizeof(float));

 	hipMemcpy(Z_d, Z, 25*25*sizeof(float), hipMemcpyHostToDevice);
 	hipMemcpy(buffer_d, buffer, 25*25*sizeof(float), hipMemcpyHostToDevice);
 	hipMemcpy(max_num_d, max_num, 25*sizeof(float), hipMemcpyHostToDevice);

 	dim3 block_size(64);
	dim3 num_of_blocks((25 * 25 + block_size.x - 1) / block_size.x);

 	softmax<<<num_of_blocks, block_size>>>(Z_d, buffer_d, max_num_d, 25, 25);
/*
 	for(i = 0; i < 10; i++){
 		for( j = 0; j < 3; ++j)
    		std::cout<<buffer[i][j]<<'\t';
	}
*/
 	hipMemcpy(max_num, max_num_d, 25*sizeof(float), hipMemcpyDeviceToHost);
 	for(i = 0; i < 25; i++){
 		std::cout<<max_num[i]<<std::endl;
 	}
 	return 0;

}