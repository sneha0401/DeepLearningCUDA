#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include <stdlib.h>
#include "softmax_activation.hh"

__global__ void softmax(float* input, float* buffer, float* max_num, int Z_x_dim, int Z_y_dim) {
  	assert(input);

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d \n", idx);
	//int col = blockIdx.x * blockDim.x + threadIdx.x;
  	// assert(input_len >= 0);  Not needed
	for (int i = 0; i < Z_y_dim; i++) {
		max_num[i] = -INFINITY;
	}

	if (idx < Z_y_dim ) {
		for (int i = 0; i < Z_x_dim; i++){ 
			if(max_num[idx] < input[idx * Z_x_dim + i] ){
				max_num[idx] = input[idx * Z_x_dim + i];
			}
			
		}
	}

	if (idx < Z_y_dim*Z_x_dim){
		for (int i = 0; i < Z_y_dim * Z_x_dim; i++){
			buffer[idx * Z_x_dim + i] = input[idx * Z_x_dim + i] - max_num[idx] ;
		}
	}
}

int main()
{
	
	float Z[25*25];
	int i ;
 	for( i = 0; i < 25*25; ++i){
		Z[i] = i+1;
    }

  	float buffer[25*25];

 	float *max_num = new float[25];

 	float *Z_d, *buffer_d, *max_num_d;
 	hipMalloc((void **)&Z_d, 25*25*sizeof(float));
 	hipMalloc((void **)&buffer_d, 25*25*sizeof(float));
 	hipMalloc((void **)&max_num_d, 25*sizeof(float));

 	hipMemcpy(Z_d, Z, 25*25*sizeof(float), hipMemcpyHostToDevice);
 	hipMemcpy(buffer_d, buffer, 25*25*sizeof(float), hipMemcpyHostToDevice);
 	hipMemcpy(max_num_d, max_num, 25*sizeof(float), hipMemcpyHostToDevice);

 	dim3 block_size(64);
	dim3 num_of_blocks((25 * 25 + block_size.x - 1) / block_size.x);

 	softmax<<<num_of_blocks, block_size>>>(Z_d, buffer_d, max_num_d, 25, 25);
	
	hipMemcpy(buffer, buffer_d, 25*25*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(max_num, max_num_d, 25*sizeof(float), hipMemcpyDeviceToHost);
 	std::cout<<"buffer"<<std::endl;
 	
	for(i = 0; i < 25*25; i++){
    	std::cout<<buffer[i]<<std::endl;
	}
	std::cout<<"max num"<<std::endl;
 	
	for(i = 0; i < 25; i++){
    	std::cout<<max_num[i]<<std::endl;
	}

 	return 0;

}