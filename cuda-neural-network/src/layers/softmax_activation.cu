#include "hip/hip_runtime.h"
#include "softmax_activation.hh"
#include "../nn_utils/nn_exception.hh"
#include <iostream>
#include <math.h>
#include <vector>


__global__ void softmaxActivationForward(float* input, float* A, float* max_num, float* row_sum, int Z_x_dim, int Z_y_dim) {
    
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	for (int i = 0; i < Z_y_dim; i++) {
		max_num[i] = -INFINITY;
	}

	if (idx < Z_y_dim ) {
		for (int i = 0; i < Z_x_dim; i++){ 
			if(max_num[idx] < input[idx * Z_x_dim + i] ){
				max_num[idx] = input[idx * Z_x_dim + i];
			}
			
		}
	}
	__syncthreads();

	if(idx < Z_x_dim){
		for(size_t i = 0; i < Z_x_dim; i++)
			A[idx * Z_y_dim + i] = expf(input[idx * Z_y_dim + i] - max_num[idx]);
	}

	if(idx < Z_x_dim){
		for(size_t i = 0; i < Z_x_dim; i++)
			row_sum[idx] += A[idx * Z_y_dim + i];
		
	}
	if(idx < Z_x_dim){
		for(size_t i = 0; i < Z_x_dim; i++)
			A[idx * Z_y_dim + i] = A[idx * Z_y_dim + i] / row_sum[idx];
	}


}
/*
void SoftmaxActivation::softmax_act(Matrix& Z){
	
	A.allocateMemoryIfNotAllocated(Z.shape.x * Z.shape.y);
	Shape shape_Y = Shape(Z.shape.y, 1);
	max_num.allocateMemoryIfNotAllocated(shape_Y);
	row_sum.allocateMemoryIfNotAllocated(shape_Y);

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);

	softmax<<<num_of_blocks, block_size>>>(Z.data_device.get(),
											A.data_device.get(),
											max_num.data_device.get(),
											row_sum.data_device.get(),
											Z.shape.x, Z.shape.y
											);
}


__global__ void softmaxActivationForward(float* Z, float* A, float* A,
										 int Z_x_dim, int Z_y_dim) {

	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < Z_x_dim * Z_y_dim) {
		A[index] = A[index];
	}
}
*/    	
  	
__global__ void softmaxActivationBackprop(float* Z, float* dA, float* dZ,
										  float* A, int Z_x_dim, int Z_y_dim) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) {
		dZ[index] = dA[index] * A[index] * (1 - A[index]);
	}
}

SoftmaxActivation::SoftmaxActivation(std::string name) {
	this->name = name;
}

SoftmaxActivation::~SoftmaxActivation()
{ }

Matrix& SoftmaxActivation::forward(Matrix& Z) {
	this->Z = Z;
	A.allocateMemoryIfNotAllocated(Z.shape);
	Shape shape_Y = Shape(Z.shape.y, 1);
	max_num.allocateMemoryIfNotAllocated(shape_Y);
	row_sum.allocateMemoryIfNotAllocated(shape_Y);

	
	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);

	softmaxActivationForward<<<num_of_blocks, block_size>>>(Z.data_device.get(),
															A.data_device.get(),
															max_num.data_device.get(),
															row_sum.data_device.get(),
															Z.shape.x, Z.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform softmax forward propagation.");

	return A;
}

Matrix& SoftmaxActivation::backprop(Matrix& dA, float learning_rate) {
	dZ.allocateMemoryIfNotAllocated(Z.shape);

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);
	softmaxActivationBackprop<<<num_of_blocks, block_size>>>(Z.data_device.get(), dA.data_device.get(),
															 dZ.data_device.get(), A.data_device.get(),
															 Z.shape.x, Z.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform softmax back propagation");

	return dZ;
}

