#include "hip/hip_runtime.h"
#include "softmax_activation.hh"
#include "../nn_utils/nn_exception.hh"
#include <iostream>
#include <math.h>
#include <vector>


__global__ void calculate_exponent_and_sum(float* value, float* sum, float* Z, int Z_x_dim, int Z_y_dim){
    // Find unique ID of each thread row and thread column
	int thread_row = blockIdx.y * blockDim.y + threadIdx.y;
	int thread_col = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize max array to store maximum of each row
    float max[Z_y_dim] = {-INFINITY};
    // Loop over the row
	for (size_t i = 0; i < Z_x_dim; i++){
    // Make sure the index doesnt exceed the number of elements in matrix
    	if(thread_row * Z_x_dim + i < Z_x_dim * Z_y_dim){
      	// If it is greater, put if in the max for the corresponding row.
    		if(Z[thread_row * Z_x_dim + i] > max[thread_row]){
      			max[thread_row] = Z[thread_row * Z_x_dim + i];
      		}
    	}
  	}
  	// Get unique index id for each thread
  	int index = thread_row * Z_x_dim + thread_col;
  	// Make sure that the thread_col is not greater than the number of rows in matrix
  	if (thread_col < Z_y_dim){
    	// Calculate exponent by subtracting each value by the max of that row
  		value[index] = expf(Z[index] - max[thread_row]);
  	}
  	// Populate sum array
  	for(size_t i = 0; i < Z_x_dim; i++){
    	// Make sure that the row ID is not greater than the number of rows in matrix
    	if(thread_row < Z_y_dim){
      	// populate each rows sum
    		sum[thread_row] += value[thread_row * Z_x_dim + i];
    	}
  	}
}


 Matrix& SoftmaxActivation::Calculate_Exponent_and_Sum(Matrix& Z){
	
	value.allocateMemoryIfNotAllocated(Z.shape.x * Z.shape.y);

	dim3 block_size(128, 128);
	dim3 num_of_blocks( (Z.shape.x + block_size.x - 1)/ block_size.x,
						(Z.shape.y + block_size.y - 1)/ block_size.y);
	calculate_exponent_and_sum<<<num_of_blocks, block_size>>>(value.data_device.get(),
														sum.data_device.get(),
														Z.data_device.get(),
														Z.shape.x, Z.shape.y,
														);
}


__global__ void softmaxActivationForward(float* Z, float* A, float* value,
										 int Z_x_dim, int Z_y_dim) {

	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < Z_x_dim * Z_y_dim) {
		A[index] = value[i];
	}
}
    	
  	
__global__ void softmaxActivationBackprop(float* Z, float* dA, float* dZ,
										  int* value, int Z_x_dim, int Z_y_dim) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) {
		dZ[index] = dA[index] * value[i] * (1 - value[i]);
	}
}

SoftmaxActivation::SoftmaxActivation(std::string name) {
	this->name = name;
}

SoftmaxActivation::~SoftmaxActivation()
{ }

Matrix& SoftmaxActivation::forward(Matrix& Z) {
	this->Z = Z;
	A.allocateMemoryIfNotAllocated(Z.shape);

	Calculate_Exponent_and_Sum(Z);

	hipDeviceSynchronize();

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);

	softmaxActivationForward<<<num_of_blocks, block_size>>>(Z.data_device.get(), A.data_device.get(),
														   	value, Z.shape.x, Z.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform softmax forward propagation.");

	return A;
}

Matrix& SoftmaxActivation::backprop(Matrix& dA, float learning_rate) {
	dZ.allocateMemoryIfNotAllocated(Z.shape);

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);
	softmaxActivationBackprop<<<num_of_blocks, block_size>>>(Z.data_device.get(), dA.data_device.get(),
															 dZ.data_device.get(), value
															 Z.shape.x, Z.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform softmax back propagation");

	return dZ;
}

